#include "hip/hip_runtime.h"
/*----------------------------------------------------------------------------*/  
/**
 *	This confidential and proprietary software may be used only as
 *	authorised by a licensing agreement from ARM Limited
 *	(C) COPYRIGHT 2011-2012 ARM Limited
 *	ALL RIGHTS RESERVED
 *
 *	The entire notice above must be reproduced on all authorised
 *	copies and copies may only be made to the extent permitted
 *	by a licensing agreement from ARM Limited.
 *
 *	@brief	Library of math functions.
 */ 
/*----------------------------------------------------------------------------*/ 

#include <time.h>
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include "mathlib.cuh"

#ifdef WIN32
	double cbrt(double n)
	{
		return n < 0 ? -pow(-n, 1.0 / 3.0) : pow(n, 1.0 / 3.0);
	}
#endif

/**************************
  basic opencl functions
**************************/

float inversesqrt(float p)
{
	return 1.0f / sqrt(p);
}
float acospi(float p)
{
	return static_cast < float >(acos(p) * (1.0f / M_PI));
};
/*float sinpi(float p)
{
	return static_cast < float >(sin(p * M_PI));
}
float cospi(float p)
{
	return static_cast < float >(cos(p * M_PI));
}*/


float nan(int p)
{
	union
	{
		int p;
		float q;
	} v;
	v.p = p | 0x7FC00000U;
	return v.q;
}

float fMax(float p, float q)
{
	if (p != p)
		return q;
	if (q != q)
		return p;
	if (p > q)
		return p;
	return q;
}

float fMin(float p, float q)
{
	if (p != p)
		return q;
	if (q != q)
		return p;
	if (p < q)
		return p;
	return q;
}


Float2 fMax(Float2 p, Float2 q)
{
	return Float2(fMax(p.x, q.x), fMax(p.y, q.y));
}

Float3 fMax(Float3 p, Float3 q)
{
	return Float3(fMax(p.x, q.x), fMax(p.y, q.y), fMax(p.z, q.z));
}

Float4 fMax(Float4 p, Float4 q)
{
	return Float4(fMax(p.x, q.x), fMax(p.y, q.y), fMax(p.z, q.z), fMax(p.w, q.w));
}


Float2 fMin(Float2 p, Float2 q)
{
	return Float2(fMin(p.x, q.x), fMin(p.y, q.y));
}

Float3 fMin(Float3 p, Float3 q)
{
	return Float3(fMin(p.x, q.x), fMin(p.y, q.y), fMin(p.z, q.z));
}

Float4 fMin(Float4 p, Float4 q)
{
	return Float4(fMin(p.x, q.x), fMin(p.y, q.y), fMin(p.z, q.z), fMin(p.w, q.w));
}

/* 
   float dot( Float2 p, Float2 q ) { return p.x*q.x + p.y*q.y; } float dot( Float3 p, Float3 q ) { return p.x*q.x + p.y*q.y + p.z*q.z; } float dot( Float4 p, Float4 q ) { return p.x*q.x + p.y*q.y +
   p.z*q.z + p.w*q.w; } */

Float3 cross(Float3 p, Float3 q)
{
	return p.yzx * q.zxy - p.zxy * q.yzx;
}

Float4 cross(Float4 p, Float4 q)
{
	return Float4(p.yzx * q.zxy - p.zxy * q.yzx, 0.0f);
}

float length(Float2 p)
{
	return sqrt(dot(p, p));
}

float length(Float3 p)
{
	return sqrt(dot(p, p));
}

float length(Float4 p)
{
	return sqrt(dot(p, p));
}

float length_sqr(Float2 p)
{
	return dot(p, p);
}

float length_sqr(Float3 p)
{
	return dot(p, p);
}

float length_sqr(Float4 p)
{
	return dot(p, p);
}


float distance(Float2 p, Float2 q)
{
	return length(q - p);
}

float distance(Float3 p, Float3 q)
{
	return length(q - p);
}

float distance(Float4 p, Float4 q)
{
	return length(q - p);
}

float distance_sqr(Float2 p, Float2 q)
{
	return length_sqr(q - p);
}

float distance_sqr(Float3 p, Float3 q)
{
	return length_sqr(q - p);
}

float distance_sqr(Float4 p, Float4 q)
{
	return length_sqr(q - p);
}


Float2 normalize(Float2 p)
{
	return p / length(p);
}

Float3 normalize(Float3 p)
{
	return p / length(p);
}

Float4 normalize(Float4 p)
{
	return p / length(p);
}


/**************************************************
  matrix functions, for 2x2, 3x3 and 4x4 matrices:

   * trace
   * determinant
   * transform
   * inverse
   * adjugate
   * characteristic polynomial
   * eigenvalue
   * eigenvector

  additionally, root solver
  for 2nd, 3rd and 4th degree monic polynomials.

*************************************************/

/* 
	struct mat2 { Float2 v[2]; };
	struct mat3 { Float3 v[3]; };
	struct mat4 { Float4 v[4]; };
*/

float trace(mat2 p)
{
	return p.v[0].x + p.v[1].y;
}

float trace(mat3 p)
{
	return p.v[0].x + p.v[1].y + p.v[2].z;
}

float trace(mat4 p)
{
	return p.v[0].x + p.v[1].y + p.v[2].z + p.v[3].w;
}

float determinant(mat2 p)
{
	Float2 v = p.v[0].xy * p.v[1].yx;
	return v.x - v.y;
}

float determinant(mat3 p)
{
	return dot(p.v[0], cross(p.v[1], p.v[2]));
}

float determinant(mat4 p)
{
	return dot(p.v[0],
			   Float4(dot(p.v[1].yzw, cross(p.v[2].yzw, p.v[3].yzw)),
					  -dot(p.v[1].xzw, cross(p.v[2].xzw, p.v[3].xzw)), dot(p.v[1].xyw, cross(p.v[2].xyw, p.v[3].xyw)), -dot(p.v[1].xyz, cross(p.v[2].xyz, p.v[3].xyz))));
}


/* 
   characteristic polynomials for matrices. These polynomials are monic, meaning that the coefficient of the highest component is 1; this component is omitted. The first component is the constant
   part. */

Float2 characteristic_poly(mat2 p)
{
	return Float2(determinant(p), -trace(p));
}


Float3 characteristic_poly(mat3 p)
{
	Float2 v1 = (p.v[0].xy * p.v[1].yx) + (p.v[0].xz * p.v[2].zx) + (p.v[1].yz * p.v[2].zy);

	return Float3(-determinant(p), v1.x - v1.y, -trace(p));
}


Float4 characteristic_poly(mat4 p)
{
	Float2 v1 = (p.v[0].xy * p.v[1].yx) + (p.v[0].xz * p.v[2].zx) + (p.v[0].xw * p.v[3].wx) + (p.v[1].yz * p.v[2].zy) + (p.v[1].yw * p.v[3].wy) + (p.v[2].zw * p.v[3].wz);

	return Float4(determinant(p),
				  -dot(p.v[1].yzw, cross(p.v[2].yzw, p.v[3].yzw))
				  - dot(p.v[0].xzw, cross(p.v[2].xzw, p.v[3].xzw)) - dot(p.v[0].xyw, cross(p.v[1].xyw, p.v[3].xyw)) - dot(p.v[0].xyz, cross(p.v[1].xyz, p.v[2].xyz)), v1.x - v1.y, -trace(p));
}


/* 
	Root finders for monic polynomials (highest coefficient is equal to 1)

	Returns a vector with length equal to the number of roots that the polynomial has;
	for roots that do not genuinely exist, we return NaN.

	The polynomial is basically

	poly(n) = p.x + p.y*n + p.z*n^2 + p.w*n^3

	(including only the components of the vector that actually exist; the next coefficient
	has the value 1, and the remaining ones have value 0. )
 */


Float2 solve_monic(Float2 p)
{
	float v = sqrt(p.y * p.y - 4 * p.x);
	return (p.yy + Float2(v, -v)) * -0.5f;
}

Float3 solve_monic(Float3 p)
{

	p = p * (1.0f / 3.0f);

	float pz = p.z;

	// compute a normalization value to scale the vector by.
	// The normalization factor is divided by 2^20.
	// This is supposed to make internal calculations unlikely
	// to overflow while also making underflows unlikely.
	float scal = 1.0f;

	float cx = static_cast < float >(cbrt(fabs(p.x)));
	float cy = static_cast < float >(cbrt(fabs(p.y)));
	scal = fMax(fMax(fabsf(p.z), cx), cy * cy) * (1.0f / 1048576.0f);
	float rscal = 1.0f / scal;
	p = p * Float3(rscal * rscal * rscal, rscal * rscal, rscal);

	float bb = p.z * p.z;		// div scal^2

	float nq = bb - p.y;		// div scal^2
	float r = 1.5f * (p.y * p.z - p.x) - p.z * bb;	// div scal^3
	float nq3 = nq * nq * nq;	// div scal^6
	float r2 = r * r;			// div scal^6

	if (nq3 < r2)
	{
		// one root
		float root = sqrt(r2 - nq3);	// div scal^3
		float s = static_cast < float >(cbrt(r + root));	// div scal
		float t = static_cast < float >(cbrt(r - root));	// div scal
		return Float3((s + t) * scal - pz, nan(0), nan(0));
	}
	else
	{
		// three roots
		float phi_r = inversesqrt(nq3);	// div scal ^ -3
		float phi_root = static_cast < float >(cbrt(phi_r * nq3));	// div scal
		float theta = acospi(r * phi_r);
		theta *= 1.0f / 3.0f;
		float ncprod = phi_root * cospi(theta);
		float dev = 1.73205080756887729353f * phi_root * sinpi(theta);
		return Float3(2 * ncprod, -dev - ncprod, dev - ncprod) * scal - pz;
	}
}


/* 
 * This function is not overflow-safe. Use with care.
 */
Float4 solve_monic(Float4 p)
{

	// step 1: depress the input polynomial
	float bias = p.w * 0.25f;
	Float3 qv = Float3((-3.0f / 256.0f) * p.w * p.w, (1.0f / 8.0f) * p.w, (-3.0 / 8.0f));
	Float3 rv = Float3((1.0f / 16.0f) * p.z * p.w - (1.0f / 4.0f) * p.y, (-1.0f / 2.0f) * p.z, 0.0f);
	Float3 qx = Float3(qv * p.w + rv) * p.w + p.xyz;

	// step 2: solve a cubic equation to get hold of a parameter p.
	Float3 monicp = Float3(-qx.y * qx.y, (qx.z * qx.z) - (4.0f * qx.x), 2.0f * qx.z);
	Float4 v = Float4(solve_monic(monicp), 1e-37f);

	// the cubic equation may have multiple solutions; at least one of them
	// is numerically at least nonnegative (but may have become negative as a result of
	// a roundoff error). We use fMax() to extract this value or a very small positive value.
	Float2 v2 = fMax(v.xy, v.zw);
	float p2 = fMax(v2.x, v2.y);	// p^2
	float pr = inversesqrt(p2);	// 1/p
	float pm = p2 * pr;			// p

	// step 3: use the solution for the cubic equation to set up two quadratic equations;
	// these two equations then result in the 4 possible roots.
	float f1 = qx.z + p2;
	float f2 = qx.y * pr;
	float s = 0.5f * (f1 + f2);
	float q = 0.5f * (f1 - f2);

	Float4 res = Float4(solve_monic(Float2(q, pm)),
						solve_monic(Float2(s, -pm)));

	// finally, order the results and apply the bias.
	if (res.x != res.x)
		return res.zwxy - bias;
	else
		return res - bias;
}



Float2 transform(mat2 p, Float2 q)
{
	return Float2(dot(p.v[0], q), dot(p.v[1], q));
}


Float3 transform(mat3 p, Float3 q)
{
	return Float3(dot(p.v[0], q), dot(p.v[1], q), dot(p.v[2], q));
}


Float4 transform(mat4 p, Float4 q)
{
	return Float4(dot(p.v[0], q), dot(p.v[1], q), dot(p.v[2], q), dot(p.v[3], q));
}



mat2 adjugate(mat2 p)
{
	mat2 res;
	res.v[0] = Float2(p.v[1].y, -p.v[0].y);
	res.v[1] = Float2(-p.v[1].x, p.v[0].x);
	return res;
}



mat2 invert(mat2 p)
{
	float rdet = 1.0f / determinant(p);
	mat2 res;
	res.v[0] = Float2(p.v[1].y, -p.v[0].y) * rdet;
	res.v[1] = Float2(-p.v[1].x, p.v[0].x) * rdet;
	return res;
}



mat3 adjugate(mat3 p)
{
	mat3 res;
	Float3 prd0 = cross(p.v[1], p.v[2]);
	Float3 prd1 = cross(p.v[2], p.v[0]);
	Float3 prd2 = cross(p.v[0], p.v[1]);
	res.v[0] = Float3(prd0.x, prd1.x, prd2.x);
	res.v[1] = Float3(prd0.y, prd1.y, prd2.y);
	res.v[2] = Float3(prd0.z, prd1.z, prd2.z);
	return res;
}



mat3 invert(mat3 p)
{
	Float3 cross0 = cross(p.v[1], p.v[2]);
	float det = dot(cross0, p.v[0]);
	float rdet = 1.0f / det;
	mat3 res;
	Float3 prd0 = cross0 * rdet;
	Float3 prd1 = cross(p.v[2], p.v[0]) * rdet;
	Float3 prd2 = cross(p.v[0], p.v[1]) * rdet;
	res.v[0] = Float3(prd0.x, prd1.x, prd2.x);
	res.v[1] = Float3(prd0.y, prd1.y, prd2.y);
	res.v[2] = Float3(prd0.z, prd1.z, prd2.z);
	return res;
}



mat4 adjugate(mat4 p)
{
	mat4 res;

	Float3 bpc0 = cross(p.v[2].yzw, p.v[3].yzw);
	Float3 tpc0 = cross(p.v[0].yzw, p.v[1].yzw);
	res.v[0] = Float4(dot(bpc0, p.v[1].yzw), -dot(bpc0, p.v[0].yzw), dot(tpc0, p.v[3].yzw), -dot(tpc0, p.v[2].yzw));

	Float3 bpc1 = cross(p.v[2].xzw, p.v[3].xzw);
	Float3 tpc1 = cross(p.v[0].xzw, p.v[1].xzw);
	res.v[1] = Float4(-dot(bpc1, p.v[1].xzw), dot(bpc1, p.v[0].xzw), -dot(tpc1, p.v[3].xzw), dot(tpc1, p.v[2].xzw));

	Float3 bpc2 = cross(p.v[2].xyw, p.v[3].xyw);
	Float3 tpc2 = cross(p.v[0].xyw, p.v[1].xyw);
	res.v[2] = Float4(dot(bpc2, p.v[1].xyw), -dot(bpc2, p.v[0].xyw), dot(tpc2, p.v[3].xyw), -dot(tpc2, p.v[2].xyw));

	Float3 bpc3 = cross(p.v[2].xyz, p.v[3].xyz);
	Float3 tpc3 = cross(p.v[0].xyz, p.v[1].xyz);
	res.v[3] = Float4(-dot(bpc3, p.v[1].xyz), dot(bpc3, p.v[0].xyz), -dot(tpc3, p.v[3].xyz), dot(tpc3, p.v[2].xyz));

	return res;
}



mat4 invert(mat4 p)
{
	// cross products between the bottom two rows
	Float3 bpc0 = cross(p.v[2].yzw, p.v[3].yzw);
	Float3 bpc1 = cross(p.v[2].xzw, p.v[3].xzw);
	Float3 bpc2 = cross(p.v[2].xyw, p.v[3].xyw);
	Float3 bpc3 = cross(p.v[2].xyz, p.v[3].xyz);

	// dot-products for the top rows
	Float4 row1 = Float4(dot(bpc0, p.v[1].yzw),
						 -dot(bpc1, p.v[1].xzw),
						 dot(bpc2, p.v[1].xyw),
						 -dot(bpc3, p.v[1].xyz));

	float det = dot(p.v[0], row1);
	float rdet = 1.0f / det;

	mat4 res;

	Float3 tpc0 = cross(p.v[0].yzw, p.v[1].yzw);
	res.v[0] = Float4(row1.x, -dot(bpc0, p.v[0].yzw), dot(tpc0, p.v[3].yzw), -dot(tpc0, p.v[2].yzw)) * rdet;

	Float3 tpc1 = cross(p.v[0].xzw, p.v[1].xzw);
	res.v[1] = Float4(row1.y, dot(bpc1, p.v[0].xzw), -dot(tpc1, p.v[3].xzw), dot(tpc1, p.v[2].xzw)) * rdet;
	Float3 tpc2 = cross(p.v[0].xyw, p.v[1].xyw);

	res.v[2] = Float4(row1.z, -dot(bpc2, p.v[0].xyw), dot(tpc2, p.v[3].xyw), -dot(tpc2, p.v[2].xyw)) * rdet;

	Float3 tpc3 = cross(p.v[0].xyz, p.v[1].xyz);
	res.v[3] = Float4(row1.w, dot(bpc3, p.v[0].xyz), -dot(tpc3, p.v[3].xyz), dot(tpc3, p.v[2].xyz)) * rdet;


	return res;
}



Float2 eigenvalues(mat2 p)
{
	return solve_monic(characteristic_poly(p));
}

Float3 eigenvalues(mat3 p)
{
	return solve_monic(characteristic_poly(p));
}

Float4 eigenvalues(mat4 p)
{
	return solve_monic(characteristic_poly(p));
}

Float2 eigenvector(mat2 p, float eigvl)
{
	// for a mat2, we first reverse-subtract the eigenvalue from the matrix diagonal,
	// then return whichever row had the larger sum-of-absolute-values.
	Float4 v = Float4(p.v[0], p.v[1]);
	v.xw = eigvl - v.xw;
	if (fabs(v.x) + fabs(v.y) > fabs(v.z) + fabs(v.w))
		return v.yx;
	else
		return v.wz;
}


Float3 eigenvector(mat3 p, float eigvl)
{
	// for a mat3, we obtain the eigenvector as follows:
	// step 1: subtract the eigenvalue from the matrix diagonal
	// step 2: take two cross products between rows in the matrix
	// step 3: return whichever of the cross products resulted in a longer vector.

	Float3 r0 = p.v[0];
	Float3 r1 = p.v[1];
	Float3 r2 = p.v[2];

	r0.x = r0.x - eigvl;
	r1.y = r1.y - eigvl;
	r2.z = r2.z - eigvl;

	Float3 v1 = cross(r0, r1);
	Float3 v2 = cross(r1, r2);

	float len1 = dot(v1, v1);
	float len2 = dot(v2, v2);
	return len1 > len2 ? v1 : v2;
}


// generalized cross product: 3 vectors with 4 components each.
// The result is a vector that is perpendicular to all the three specified vectors.

// it works in the sense that it produces a perpendicular-to-everyting vector,
// but it has not been tested whether it points in the "right" direction.
Float4 gcross(Float4 p, Float4 q, Float4 r)
{
	return Float4(dot(p.yzw, cross(q.yzw, r.yzw)), -dot(p.xzw, cross(q.xzw, r.xzw)), dot(p.xyw, cross(q.xyw, r.xyw)), -dot(p.xyz, cross(q.xyz, r.xyz)));
}



Float4 eigenvector(mat4 p, float eigvl)
{
	Float4 r0 = p.v[0];
	Float4 r1 = p.v[1];
	Float4 r2 = p.v[2];
	Float4 r3 = p.v[3];

	r0.x = r0.x - eigvl;
	r1.y = r1.y - eigvl;
	r2.z = r2.z - eigvl;
	r3.w = r3.w - eigvl;

	// generate four candidate vectors using the generalized cross product.
	// These will in general point in the same direction (or 180 degree opposite),
	// however they will have different lengths. Pick the longest one.
	Float3 tpc0 = cross(r0.yzw, r1.yzw);
	Float3 tpc1 = cross(r0.xzw, r1.xzw);
	Float3 tpc2 = cross(r0.xyw, r1.xyw);
	Float3 tpc3 = cross(r0.xyz, r1.xyz);

	Float4 v1 = Float4(dot(r2.yzw, tpc0),
					   -dot(r2.xzw, tpc1),
					   dot(r2.xyw, tpc2),
					   -dot(r2.xyz, tpc3));

	Float4 v2 = Float4(dot(r3.yzw, tpc0),
					   -dot(r3.xzw, tpc1),
					   dot(r3.xyw, tpc2),
					   -dot(r3.xyz, tpc3));

	Float3 bpc0 = cross(r2.yzw, r3.yzw);
	Float3 bpc1 = cross(r2.xzw, r3.xzw);
	Float3 bpc2 = cross(r2.xyw, r3.xyw);
	Float3 bpc3 = cross(r2.xyz, r3.xyz);

	Float4 v3 = Float4(dot(r0.yzw, bpc0),
					   -dot(r0.xzw, bpc1),
					   dot(r0.xyw, bpc2),
					   -dot(r0.xyz, bpc3));

	Float4 v4 = Float4(dot(r1.yzw, bpc0),
					   -dot(r1.xzw, bpc1),
					   dot(r1.xyw, bpc2),
					   -dot(r1.xyz, bpc3));

	float len1 = dot(v1, v1);
	float len2 = dot(v2, v2);
	float len3 = dot(v3, v3);
	float len4 = dot(v4, v4);

	if (fMax(len1, len2) > fMax(len3, len4))
		return len1 > len2 ? v1 : v2;
	else
		return len3 > len4 ? v3 : v4;
}


// matrix multiply

mat2 operator *(mat2 a, mat2 b)
{
	mat2 res;
	res.v[0] = a.v[0].x * b.v[0] + a.v[0].y * b.v[1];
	res.v[1] = a.v[1].x * b.v[0] + a.v[1].y * b.v[1];
	return res;
}

mat3 operator *(mat3 a, mat3 b)
{
	mat3 res;
	res.v[0] = a.v[0].x * b.v[0] + a.v[0].y * b.v[1] + a.v[0].z * b.v[2];
	res.v[1] = a.v[1].x * b.v[0] + a.v[1].y * b.v[1] + a.v[1].z * b.v[2];
	res.v[2] = a.v[2].x * b.v[0] + a.v[2].y * b.v[1] + a.v[2].z * b.v[2];
	return res;
}

mat4 operator *(mat4 a, mat4 b)
{
	mat4 res;
	res.v[0] = a.v[0].x * b.v[0] + a.v[0].y * b.v[1] + a.v[0].z * b.v[2] + a.v[0].w * b.v[3];
	res.v[1] = a.v[1].x * b.v[0] + a.v[1].y * b.v[1] + a.v[1].z * b.v[2] + a.v[1].w * b.v[3];
	res.v[2] = a.v[2].x * b.v[0] + a.v[2].y * b.v[1] + a.v[2].z * b.v[2] + a.v[2].w * b.v[3];
	res.v[3] = a.v[3].x * b.v[0] + a.v[3].y * b.v[1] + a.v[3].z * b.v[2] + a.v[3].w * b.v[3];
	return res;
}



/*************************

simple geometric functions

*************************/


// return parameter value for the point on the line closest to the specified point
float param_nearest_on_line(Float2 point, line2 line)
{
	return dot(point - line.a, line.b) / dot(line.b, line.b);
}

float param_nearest_on_line(Float3 point, line3 line)
{
	return dot(point - line.a, line.b) / dot(line.b, line.b);
}

float param_nearest_on_line(Float4 point, line4 line)
{
	return dot(point - line.a, line.b) / dot(line.b, line.b);
}


// return distance between point and line
float point_line_distance(Float2 point, line2 line)
{
	return distance(point, line.a + line.b * param_nearest_on_line(point, line));
}

float point_line_distance(Float3 point, line3 line)
{
	return distance(point, line.a + line.b * param_nearest_on_line(point, line));
}

float point_line_distance(Float4 point, line4 line)
{
	return distance(point, line.a + line.b * param_nearest_on_line(point, line));
}


float point_line_distance_sqr(Float2 point, line2 line)
{
	return distance_sqr(point, line.a + line.b * param_nearest_on_line(point, line));
}

float point_line_distance_sqr(Float3 point, line3 line)
{
	return distance_sqr(point, line.a + line.b * param_nearest_on_line(point, line));
}

float point_line_distance_sqr(Float4 point, line4 line)
{
	return distance_sqr(point, line.a + line.b * param_nearest_on_line(point, line));
}



// distance between plane/hyperplane in 3D and 4D
float point_plane_3d_distance(Float3 point, plane_3d plane)
{
	return dot(point - plane.root_point, plane.normal);
}


float point_hyperplane_4d_distance(Float4 point, hyperplane_4d plane)
{
	return dot(point - plane.root_point, plane.normal);
}


// helper functions to produce a 3D plane from three points and a 4D hyperplane from four points.
plane_3d generate_plane_from_points(Float3 point0, Float3 point1, Float3 poInt2)
{
	plane_3d res;
	res.root_point = point0;
	res.normal = normalize(cross(point1 - point0, poInt2 - point0));
	return res;
}

hyperplane_4d generate_hyperplane_from_points(Float4 point0, Float4 point1, Float4 poInt2, Float4 poInt3)
{
	hyperplane_4d res;
	res.root_point = point0;
	res.normal = normalize(gcross(point1 - point0, poInt2 - point0, poInt3 - point0));
	return res;
}


